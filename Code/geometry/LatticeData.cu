#include "hip/hip_runtime.h"

// This file is part of HemeLB and is Copyright (C)
// the HemeLB team and/or their institutions, as detailed in the
// file AUTHORS. This software is provided under the terms of the
// license in the file LICENSE.

#include "geometry/LatticeData.h"



using namespace hemelb;
using namespace hemelb::geometry;



__global__
void CopyReceivedKernel(
  const site_t* streamingIndicesForReceivedDistributions,
  const distribn_t* fOldShared,
  distribn_t* fNew,
  site_t totalSharedFs
)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if ( i >= totalSharedFs )
  {
    return;
  }

  fNew[streamingIndicesForReceivedDistributions[i]] = fOldShared[i];
}



void LatticeData::CopyReceivedGPU(int blockSize)
{
  if ( totalSharedFs == 0 )
  {
    return;
  }

  int gridSize = (totalSharedFs + blockSize - 1) / blockSize;

  CopyReceivedKernel<<<gridSize, blockSize>>>(
    streamingIndicesForReceivedDistributions_dev,
    GetFOldGPU(neighbouringProcs[0].FirstSharedDistribution),
    GetFNewGPU(0),
    totalSharedFs
  );
  CUDA_SAFE_CALL(hipGetLastError());
}

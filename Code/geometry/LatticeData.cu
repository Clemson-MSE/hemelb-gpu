
#include <hip/hip_runtime.h>

// units.h
typedef int64_t site_t;
typedef double distribn_t;



namespace hemelb {
namespace geometry {



__global__ void LatticeData_CopyReceived(
  const site_t* streamingIndicesForReceivedDistributions,
  const distribn_t* fOldShared,
  distribn_t* fNew,
  site_t totalSharedFs
)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if ( i >= totalSharedFs )
  {
    return;
  }

  fNew[streamingIndicesForReceivedDistributions[i]] = fOldShared[i];
}



__host__ void LatticeData_CopyReceivedGPU(
  const site_t* streamingIndicesForReceivedDistributions,
  const distribn_t* fOldShared,
  distribn_t* fNew,
  site_t totalSharedFs
)
{
  const int BLOCK_SIZE = 256;
  const int GRID_SIZE = (totalSharedFs + BLOCK_SIZE - 1) / BLOCK_SIZE;

  LatticeData_CopyReceived<<<GRID_SIZE, BLOCK_SIZE>>>(
    streamingIndicesForReceivedDistributions,
    fOldShared,
    fNew,
    totalSharedFs
  );
}



}
}


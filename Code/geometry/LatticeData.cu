#include "hip/hip_runtime.h"

// This file is part of HemeLB and is Copyright (C)
// the HemeLB team and/or their institutions, as detailed in the
// file AUTHORS. This software is provided under the terms of the
// license in the file LICENSE.

#include "geometry/LatticeData.h"



using namespace hemelb;
using namespace hemelb::geometry;



__global__
void CopyReceivedKernel(
  const site_t* streamingIndicesForReceivedDistributions,
  const distribn_t* fOldShared,
  distribn_t* fNew,
  site_t totalSharedFs
)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if ( i >= totalSharedFs )
  {
    return;
  }

  fNew[streamingIndicesForReceivedDistributions[i]] = fOldShared[i];
}



void LatticeData::CopyReceivedGPU()
{
  if ( totalSharedFs == 0 )
  {
    return;
  }

  const int BLOCK_SIZE = 256;
  const int GRID_SIZE = (totalSharedFs + BLOCK_SIZE - 1) / BLOCK_SIZE;

  CopyReceivedKernel<<<GRID_SIZE, BLOCK_SIZE>>>(
    streamingIndicesForReceivedDistributions_dev,
    GetFOldGPU(neighbouringProcs[0].FirstSharedDistribution),
    GetFNewGPU(0),
    totalSharedFs
  );
  CUDA_SAFE_CALL(hipGetLastError());
}

#include "hip/hip_runtime.h"

// This file is part of HemeLB and is Copyright (C)
// the HemeLB team and/or their institutions, as detailed in the
// file AUTHORS. This software is provided under the terms of the
// license in the file LICENSE.

#include "units.h"



namespace hemelb {
namespace geometry {



__global__ void LatticeData_CopyReceived(
  const site_t* streamingIndicesForReceivedDistributions,
  const distribn_t* fOldShared,
  distribn_t* fNew,
  site_t totalSharedFs
)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if ( i >= totalSharedFs )
  {
    return;
  }

  fNew[streamingIndicesForReceivedDistributions[i]] = fOldShared[i];
}



__host__ void LatticeData_CopyReceivedGPU(
  const site_t* streamingIndicesForReceivedDistributions,
  const distribn_t* fOldShared,
  distribn_t* fNew,
  site_t totalSharedFs
)
{
  const int BLOCK_SIZE = 256;
  const int GRID_SIZE = (totalSharedFs + BLOCK_SIZE - 1) / BLOCK_SIZE;

  LatticeData_CopyReceived<<<GRID_SIZE, BLOCK_SIZE>>>(
    streamingIndicesForReceivedDistributions,
    fOldShared,
    fNew,
    totalSharedFs
  );
}



}
}

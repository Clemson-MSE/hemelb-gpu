
#include <hip/hip_runtime.h>

// units.h
typedef int64_t site_t;
typedef double distribn_t;



// geometry/SiteType.h
enum site_type_t
{
  SOLID_TYPE = 0U,
  FLUID_TYPE = 1U,
  INLET_TYPE = 2U,
  OUTLET_TYPE = 3U
};



// geometry/SiteDataBare.h
typedef struct
{
  unsigned wallIntersection;
  unsigned ioletIntersection;
  site_type_t type;
  int ioletId;
} site_data_t;



__device__ bool Site_HasIolet(unsigned ioletIntersection, int direction)
{
  unsigned mask = 1U << (direction - 1);
  return ((ioletIntersection & mask) != 0) && (direction > 0);
}



__device__ bool Site_HasWall(unsigned wallIntersection, int direction)
{
  unsigned mask = 1U << (direction - 1);
  return ((wallIntersection & mask) != 0) && (direction > 0);
}



// lb/iolets/InOutLetCosine.h
typedef struct
{
  distribn_t minimumSimulationDensity;
  double3 normal;
  double densityMean;
  double densityAmp;
  double phase;
  double period;
  unsigned int warmUpLength;
} iolet_cosine_t;



__device__ distribn_t InOutLetCosine_GetDensity(const iolet_cosine_t& iolet, unsigned long timeStep)
{
  distribn_t w = 2.0 * M_PI / iolet.period;

  distribn_t target = iolet.densityMean + iolet.densityAmp * cos(w * timeStep + iolet.phase);

  if (timeStep >= iolet.warmUpLength)
  {
    return target;
  }

  double interpolationFactor = ((double) timeStep) / ((double) iolet.warmUpLength);

  return interpolationFactor * target + (1. - interpolationFactor) * iolet.minimumSimulationDensity;
}



// lb/lattices/D3Q15.h
namespace D3Q15
{
  __constant__ const int NUMVECTORS = 15;

  __constant__ const distribn_t CXD[] = { 0, 1, -1, 0, 0, 0, 0, 1, -1, 1, -1, 1, -1, 1, -1 };
  __constant__ const distribn_t CYD[] = { 0, 0, 0, 1, -1, 0, 0, 1, -1, 1, -1, -1, 1, -1, 1 };
  __constant__ const distribn_t CZD[] = { 0, 0, 0, 0, 0, 1, -1, 1, -1, -1, 1, 1, -1, -1, 1 };

  __constant__ const distribn_t EQMWEIGHTS[] = {
    2.0 / 9.0,
    1.0 / 9.0,
    1.0 / 9.0,
    1.0 / 9.0,
    1.0 / 9.0,
    1.0 / 9.0,
    1.0 / 9.0,
    1.0 / 72.0,
    1.0 / 72.0,
    1.0 / 72.0,
    1.0 / 72.0,
    1.0 / 72.0,
    1.0 / 72.0,
    1.0 / 72.0,
    1.0 / 72.0
  };

  __constant__ const int INVERSEDIRECTIONS[] = { 0, 2, 1, 4, 3, 6, 5, 8, 7, 10, 9, 12, 11, 14, 13 };
}


// lb/lattices/Lattice.h
__device__ void Lattice_CalculateFeq(const distribn_t& density, const double3& momentum, distribn_t* f_eq)
{
  const distribn_t density_1 = 1. / density;
  const distribn_t momentumMagnitudeSquared =
      momentum.x * momentum.x
      + momentum.y * momentum.y
      + momentum.z * momentum.z;

  for ( int j = 0; j < D3Q15::NUMVECTORS; ++j )
  {
    const distribn_t mom_dot_ei =
        D3Q15::CXD[j] * momentum.x
        + D3Q15::CYD[j] * momentum.y
        + D3Q15::CZD[j] * momentum.z;

    f_eq[j] = D3Q15::EQMWEIGHTS[j]
        * (density
            - (3. / 2.) * momentumMagnitudeSquared * density_1
            + (9. / 2.) * density_1 * mom_dot_ei * mom_dot_ei
            + 3. * mom_dot_ei);
  }
}



__global__ void DoStreamAndCollideKernel(
  site_t firstIndex,
  site_t siteCount,
  distribn_t lbmParams_tau,
  distribn_t lbmParams_omega,
  const iolet_cosine_t* inlets,
  const iolet_cosine_t* outlets,
  const site_t* neighbourIndices,
  const site_data_t* siteData,
  const distribn_t* fOld,
  distribn_t* fNew,
  unsigned long timeStep
)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if ( i >= siteCount )
  {
    return;
  }

  site_t siteIndex = firstIndex + i;

  // initialize hydroVars
  distribn_t f[D3Q15::NUMVECTORS];
  distribn_t density;
  double3 momentum;
  double3 velocity;
  distribn_t f_eq[D3Q15::NUMVECTORS];
  distribn_t* f_neq = f_eq;
  distribn_t* f_post = f_eq;

  // copy fOld to local memory
  memcpy(&f[0], &fOld[siteIndex * D3Q15::NUMVECTORS], D3Q15::NUMVECTORS * sizeof(distribn_t));

  // collider.CalculatePreCollision() (collider = Normal, kernel = LBGK)

  // Lattice::CalculateDensityMomentumFEq()
  density = 0.0;
  momentum.x = 0.0;
  momentum.y = 0.0;
  momentum.z = 0.0;

  for ( int j = 0; j < D3Q15::NUMVECTORS; ++j )
  {
    density += f[j];
    momentum.x += D3Q15::CXD[j] * f[j];
    momentum.y += D3Q15::CYD[j] * f[j];
    momentum.z += D3Q15::CZD[j] * f[j];
  }

  velocity.x = momentum.x / density;
  velocity.y = momentum.y / density;
  velocity.z = momentum.z / density;

  Lattice_CalculateFeq(density, momentum, f_eq);

  // LBGK::DoCalculateDensityMomentumFeq()
  for ( int j = 0; j < D3Q15::NUMVECTORS; ++j )
  {
    f_neq[j] = f[j] - f_eq[j];
  }

  // collider.Collide()

  // LBGK::DoCollide()
  for ( int j = 0; j < D3Q15::NUMVECTORS; ++j )
  {
    f_post[j] = f[j] + f_neq[j] * lbmParams_omega;
  }

  // perform streaming
  site_data_t site = siteData[siteIndex];

  for ( int j = 0; j < D3Q15::NUMVECTORS; ++j )
  {
    if ( Site_HasIolet(site.ioletIntersection, j) )
    {
      // get iolet
      iolet_cosine_t iolet = (site.type == INLET_TYPE)
        ? inlets[site.ioletId]
        : outlets[site.ioletId];

      // get density at the iolet
      distribn_t ghost_density = InOutLetCosine_GetDensity(iolet, timeStep);

      // compute momentum at the iolet
      distribn_t component =
          velocity.x * iolet.normal.x
          + velocity.y * iolet.normal.y
          + velocity.z * iolet.normal.z;

      double3 ghost_momentum;
      ghost_momentum.x = iolet.normal.x * component * ghost_density;
      ghost_momentum.y = iolet.normal.y * component * ghost_density;
      ghost_momentum.z = iolet.normal.z * component * ghost_density;

      // compute f_eq at the iolet
      distribn_t ghost_f_eq[D3Q15::NUMVECTORS];

      Lattice_CalculateFeq(ghost_density, ghost_momentum, ghost_f_eq);

      int outIndex = siteIndex * D3Q15::NUMVECTORS + D3Q15::INVERSEDIRECTIONS[j];
      fNew[outIndex] = ghost_f_eq[D3Q15::INVERSEDIRECTIONS[j]];
    }
    else if ( Site_HasWall(site.wallIntersection, j) )
    {
      int outIndex = siteIndex * D3Q15::NUMVECTORS + D3Q15::INVERSEDIRECTIONS[j];
      fNew[outIndex] = f_post[j];
    }
    else
    {
      int outIndex = neighbourIndices[siteIndex * D3Q15::NUMVECTORS + j];
      fNew[outIndex] = f_post[j];
    }
  }
}



namespace hemelb {
namespace lb {
namespace streamers {



__host__ void DoStreamAndCollideGPU(
  site_t firstIndex,
  site_t siteCount,
  distribn_t lbmParams_tau,
  distribn_t lbmParams_omega,
  const iolet_cosine_t* inlets,
  const iolet_cosine_t* outlets,
  const site_t* neighbourIndices,
  const void* siteData,
  const distribn_t* fOld,
  distribn_t* fNew,
  unsigned long timeStep
)
{
  const int BLOCK_SIZE = 256;
  const int GRID_SIZE = (siteCount + BLOCK_SIZE - 1) / BLOCK_SIZE;

  DoStreamAndCollideKernel<<<GRID_SIZE, BLOCK_SIZE>>>(
    firstIndex,
    siteCount,
    lbmParams_tau,
    lbmParams_omega,
    inlets,
    outlets,
    neighbourIndices,
    (site_data_t*) siteData,
    fOld,
    fNew,
    timeStep
  );
}



}
}
}

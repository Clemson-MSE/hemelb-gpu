#include "hip/hip_runtime.h"

// This file is part of HemeLB and is Copyright (C)
// the HemeLB team and/or their institutions, as detailed in the
// file AUTHORS. This software is provided under the terms of the
// license in the file LICENSE.

#include "lb/collisions/Collisions.h"
#include "lb/iolets/InOutLetCosine.cuh"
#include "lb/kernels/Kernels.h"
#include "lb/lattices/Lattices.h"
#include "lb/streamers/Streamers.h"

#include "lb/lattices/D3Q15.cuh"
#include "lb/lattices/D3Q19.cuh"
#include "lb/lattices/D3Q27.cuh"



using namespace hemelb;
using namespace hemelb::lb;



#define DmQn lattices::GPU:: HEMELB_LATTICE



class Normal_LBGK_SBB_Nash
{
public:
  typedef lattices:: HEMELB_LATTICE LatticeType;
  typedef typename collisions::Normal<kernels::LBGK<LatticeType>> CollisionType;
  typedef typename streamers::SimpleBounceBackDelegate<CollisionType> WallLinkType;
  typedef typename streamers::NashZerothOrderPressureDelegate<CollisionType> IoletLinkType;

  typedef typename streamers::StreamerTypeFactory<
    CollisionType,
    WallLinkType,
    IoletLinkType
  > Type;
};



__device__
int Normal_LBGK_SBB_Nash_GetOutputIndex(
  site_t siteIndex,
  Direction direction,
  const geometry::SiteData& site,
  const site_t* neighbourIndices
)
{
  // NashZerothOrderPressureDelegate::StreamLink()
  // SimpleBounceBackDelegate::StreamLink()
  // SimpleCollideAndStreamDelegate::StreamLink()
  return (site.HasIolet(direction) || site.HasWall(direction))
    ? siteIndex * DmQn::NUMVECTORS + DmQn::INVERSEDIRECTIONS[direction]
    : neighbourIndices[siteIndex * DmQn::NUMVECTORS + direction];
}



__global__
void Normal_LBGK_SBB_Nash_StreamAndCollide(
  site_t firstIndex,
  site_t siteCount,
  distribn_t lbmParams_tau,
  distribn_t lbmParams_omega,
  const iolets::InOutLetCosineGPU* inlets,
  const iolets::InOutLetCosineGPU* outlets,
  const site_t* neighbourIndices,
  const geometry::SiteData* siteData,
  const distribn_t* fOld,
  distribn_t* fNew,
  unsigned long timeStep
)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if ( i >= siteCount )
  {
    return;
  }

  site_t siteIndex = firstIndex + i;
  auto& site = siteData[siteIndex];

  // initialize hydroVars
  distribn_t f[DmQn::NUMVECTORS];
  distribn_t density = 0.0;
  double3 momentum = make_double3(0.0, 0.0, 0.0);
  distribn_t f_post[DmQn::NUMVECTORS];

  for ( Direction j = 0; j < DmQn::NUMVECTORS; ++j )
  {
    // copy fOld to local memory
    f[j] = fOld[siteIndex * DmQn::NUMVECTORS + j];

    // Normal::DoCalculatePreCollision()
    // LBGK::DoCalculateDensityMomentumFeq()
    // Lattice::CalculateDensityAndMomentum()
    density += f[j];
    momentum.x += DmQn::CXD[j] * f[j];
    momentum.y += DmQn::CYD[j] * f[j];
    momentum.z += DmQn::CZD[j] * f[j];
  }

  // Lattice::CalculateFeq()
  const distribn_t density_1 = 1. / density;
  const distribn_t momentumMagnitudeSquared =
      momentum.x * momentum.x
      + momentum.y * momentum.y
      + momentum.z * momentum.z;

  for ( Direction j = 0; j < DmQn::NUMVECTORS; ++j )
  {
    if ( site.HasIolet(j) )
    {
      // NashZerothOrderPressureDelegate::StreamLink()
      // get iolet
      auto& iolet = (site.GetSiteType() == geometry::INLET_TYPE)
        ? inlets[site.GetIoletId()]
        : outlets[site.GetIoletId()];

      // get density at the iolet
      distribn_t ioletDensity = iolet.GetDensity(timeStep);

      // compute momentum at the iolet
      distribn_t component =
          (momentum.x / density) * iolet.normal.x
          + (momentum.y / density) * iolet.normal.y
          + (momentum.z / density) * iolet.normal.z;

      double3 ioletMomentum;
      ioletMomentum.x = iolet.normal.x * component * ioletDensity;
      ioletMomentum.y = iolet.normal.y * component * ioletDensity;
      ioletMomentum.z = iolet.normal.z * component * ioletDensity;

      // compute f_eq at the iolet
      // Lattice::CalculateFeq()
      const distribn_t ioletDensity_1 = 1. / ioletDensity;
      const distribn_t momentumMagnitudeSquared =
          ioletMomentum.x * ioletMomentum.x
          + ioletMomentum.y * ioletMomentum.y
          + ioletMomentum.z * ioletMomentum.z;

      Direction jj = DmQn::INVERSEDIRECTIONS[j];
      const distribn_t mom_dot_ei =
          DmQn::CXD[jj] * ioletMomentum.x
          + DmQn::CYD[jj] * ioletMomentum.y
          + DmQn::CZD[jj] * ioletMomentum.z;

      f_post[j] = DmQn::EQMWEIGHTS[jj]
          * (ioletDensity
              - (3. / 2.) * ioletDensity_1 * momentumMagnitudeSquared
              + (9. / 2.) * ioletDensity_1 * mom_dot_ei * mom_dot_ei
              + 3. * mom_dot_ei);
    }
    else
    {
      // Lattice::CalculateFeq()
      const distribn_t mom_dot_ei =
          DmQn::CXD[j] * momentum.x
          + DmQn::CYD[j] * momentum.y
          + DmQn::CZD[j] * momentum.z;

      f_post[j] = DmQn::EQMWEIGHTS[j]
          * (density
              - (3. / 2.) * density_1 * momentumMagnitudeSquared
              + (9. / 2.) * density_1 * mom_dot_ei * mom_dot_ei
              + 3. * mom_dot_ei);

      // Normal::DoCollide()
      // LBGK::DoCollide()
      f_post[j] = f[j] + lbmParams_omega * (f[j] - f_post[j]);
    }

    // perform streaming
    int outIndex = Normal_LBGK_SBB_Nash_GetOutputIndex(siteIndex, j, site, neighbourIndices);

    fNew[outIndex] = f_post[j];
  }
}



template<>
void Normal_LBGK_SBB_Nash::Type::StreamAndCollideGPU(
  const site_t firstIndex,
  const site_t siteCount,
  const lb::LbmParameters* lbmParams,
  geometry::LatticeData* latDat,
  lb::SimulationState* simState,
  const iolets::InOutLetCosineGPU* inlets,
  const iolets::InOutLetCosineGPU* outlets,
  int blockSize
)
{
  if ( siteCount == 0 )
  {
    return;
  }

  const int GRID_SIZE = (siteCount + blockSize - 1) / blockSize;

  Normal_LBGK_SBB_Nash_StreamAndCollide<<<GRID_SIZE, blockSize>>>(
    firstIndex,
    siteCount,
    lbmParams->GetTau(),
    lbmParams->GetOmega(),
    inlets,
    outlets,
    latDat->GetNeighbourIndicesGPU(),
    latDat->GetSiteDataGPU(),
    latDat->GetFOldGPU(0),
    latDat->GetFNewGPU(0),
    simState->Get0IndexedTimeStep()
  );
  CUDA_SAFE_CALL(hipGetLastError());
}

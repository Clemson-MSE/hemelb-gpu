#include "hip/hip_runtime.h"
#include "lb/lattices/Lattices.h"
#include "lb/streamers/StreamerTypeFactory.cuh"
#include "iostream"

#define CUDA_SAFE_CALL(x)                           \
{                                                   \
    hipError_t error = x;                          \
    if ( error != hipSuccess ) {                   \
      const char *name = hipGetErrorName(error);   \
      const char *str = hipGetErrorString(error);  \
      std::cerr << "\n"                             \
                << "CUDA Error at " #x "\n"         \
                << name << ": " << str << "\n";     \
      exit(1);                                      \
    }                                               \
}


namespace hemelb {
namespace lb {
namespace streamers {

const int MAX_LATTICE_NUMVECTORS=27;
__constant__ Direction d_NUMVECTORS = MAX_LATTICE_NUMVECTORS;
__constant__ int d_CX[MAX_LATTICE_NUMVECTORS];
__constant__ int d_CY[MAX_LATTICE_NUMVECTORS];
__constant__ int d_CZ[MAX_LATTICE_NUMVECTORS];
__constant__ int* d_discreteVelocityVectors[3] = {d_CX, d_CY, d_CZ};
__constant__ distribn_t d_CXD[MAX_LATTICE_NUMVECTORS];
__constant__ distribn_t d_CYD[MAX_LATTICE_NUMVECTORS];
__constant__ distribn_t d_CZD[MAX_LATTICE_NUMVECTORS];
__constant__ distribn_t d_EQMWEIGHTS[MAX_LATTICE_NUMVECTORS];
__constant__ Direction d_INVERSEDIRECTIONS[MAX_LATTICE_NUMVECTORS];

template <typename LatticeType>
void FillGPUConstantMemory() {
Direction num_vectors = LatticeType::NUMVECTORS;
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_NUMVECTORS), &num_vectors, sizeof(Direction)));
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_CX), LatticeType::CX, LatticeType::NUMVECTORS * sizeof(int)));
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_CY), LatticeType::CY, LatticeType::NUMVECTORS * sizeof(int)));
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_CZ), LatticeType::CZ, LatticeType::NUMVECTORS * sizeof(int)));
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_discreteVelocityVectors), LatticeType::discreteVelocityVectors, LatticeType::NUMVECTORS * 3 *sizeof(int)));
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_CXD), LatticeType::CXD, LatticeType::NUMVECTORS * sizeof(distribn_t)));
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_CYD), LatticeType::CYD, LatticeType::NUMVECTORS * sizeof(distribn_t)));
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_CZD), LatticeType::CZD, LatticeType::NUMVECTORS * sizeof(distribn_t)));
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_EQMWEIGHTS), LatticeType::EQMWEIGHTS, LatticeType::NUMVECTORS * sizeof(distribn_t)));
CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_INVERSEDIRECTIONS), LatticeType::INVERSEDIRECTIONS, LatticeType::NUMVECTORS * sizeof(Direction)));
}

template void FillGPUConstantMemory<hemelb::lb::lattices::D3Q15>();
template void FillGPUConstantMemory<hemelb::lb::lattices::D3Q15i>();
template void FillGPUConstantMemory<hemelb::lb::lattices::D3Q19>();
template void FillGPUConstantMemory<hemelb::lb::lattices::D3Q27>();

// type definitions taken from host code
typedef int64_t site_t;
typedef double distribn_t;

__device__ bool Site_HasWall(unsigned wallIntersection, int direction)
{
  unsigned mask = 1U << max(0, direction - 1);
  return (wallIntersection & mask) != 0;
}

__global__ void DoStreamAndCollideKernel(
  site_t firstIndex,
  site_t siteCount,
  distribn_t lbmParams_tau,
  distribn_t lbmParams_omega,
  const site_t* neighbourIndices,
  const unsigned* wallIntersections,
  const distribn_t* fOld,
  distribn_t* fNew
)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if ( i >= siteCount )
  {
    return;
  }

  site_t siteIndex = firstIndex + i;

  // initialize hydroVars
  distribn_t f[MAX_LATTICE_NUMVECTORS];
  distribn_t density;
  double3 momentum;
  double3 velocity;
  distribn_t f_eq[MAX_LATTICE_NUMVECTORS];
  distribn_t* f_neq = f_eq;
  distribn_t* f_post = f_eq;

  // copy fOld to local memory
  memcpy(&f[0], &fOld[siteIndex * d_NUMVECTORS], d_NUMVECTORS * sizeof(distribn_t));

  // collider.CalculatePreCollision() (collider = Normal, kernel = LBGK)

  // Lattice::CalculateDensityMomentumFEq()
  density = 0.0;
  momentum.x = 0.0;
  momentum.y = 0.0;
  momentum.z = 0.0;

  for ( int j = 0; j < d_NUMVECTORS; ++j )
  {
    density += f[j];
    momentum.x += d_CXD[j] * f[j];
    momentum.y += d_CYD[j] * f[j];
    momentum.z += d_CZD[j] * f[j];
  }

  velocity.x = momentum.x / density;
  velocity.y = momentum.y / density;
  velocity.z = momentum.z / density;

  const distribn_t density_1 = 1. / density;
  const distribn_t momentumMagnitudeSquared =
      momentum.x * momentum.x
      + momentum.y * momentum.y
      + momentum.z * momentum.z;

  for ( int j = 0; j < d_NUMVECTORS; ++j )
  {
    const distribn_t mom_dot_ei =
        d_CX[j] * momentum.x
        + d_CY[j] * momentum.y
        + d_CZ[j] * momentum.z;

    f_eq[j] = d_EQMWEIGHTS[j]
        * (density
            - (3. / 2.) * momentumMagnitudeSquared * density_1
            + (9. / 2.) * density_1 * mom_dot_ei * mom_dot_ei
            + 3. * mom_dot_ei);
  }

  // LBGK::DoCalculateDensityMomentumFeq()
  for ( int j = 0; j < d_NUMVECTORS; ++j )
  {
    f_neq[j] = f[j] - f_eq[j];
  }

  // collider.Collide()

  // LBGK::DoCollide()
  for ( int j = 0; j < d_NUMVECTORS; ++j )
  {
    f_post[j] = f[j] + f_neq[j] * lbmParams_omega;
  }

  // perform streaming
  for ( int j = 0; j < d_NUMVECTORS; ++j )
  {
    if ( Site_HasWall(wallIntersections[siteIndex], j) )
    {
      int outIndex = siteIndex * d_NUMVECTORS + d_INVERSEDIRECTIONS[j];
      fNew[outIndex] = f_post[j];
    }
    else
    {
      int outIndex = neighbourIndices[siteIndex * d_NUMVECTORS + j];
      fNew[outIndex] = f_post[j];
    }
  }
}

void DoStreamAndCollideGPU(
  site_t firstIndex,
  site_t siteCount,
  distribn_t lbmParams_tau,
  distribn_t lbmParams_omega,
  const site_t* neighbourIndices,
  const unsigned* wallIntersections,
  const distribn_t* fOld,
  distribn_t* fNew
)
{
  const int BLOCK_SIZE = 256;
  const int GRID_SIZE = (siteCount + BLOCK_SIZE - 1) / BLOCK_SIZE;

  DoStreamAndCollideKernel <<<GRID_SIZE, BLOCK_SIZE>>>(
    firstIndex,
    siteCount,
    lbmParams_tau,
    lbmParams_omega,
    neighbourIndices,
    wallIntersections,
    fOld,
    fNew
  );
}

}
}
}

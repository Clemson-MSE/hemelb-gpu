#include "hip/hip_runtime.h"

// This file is part of HemeLB and is Copyright (C)
// the HemeLB team and/or their institutions, as detailed in the
// file AUTHORS. This software is provided under the terms of the
// license in the file LICENSE.

#include "lb/collisions/Collisions.h"
#include "lb/iolets/InOutLetCosine.cuh"
#include "lb/kernels/Kernels.h"
#include "lb/lattices/Lattices.h"
#include "lb/streamers/Streamers.h"

#include "lb/lattices/D3Q15.cuh"
#include "lb/lattices/D3Q19.cuh"
#include "lb/lattices/D3Q27.cuh"



using namespace hemelb;
using namespace hemelb::lb;



#define DmQn lattices::GPU:: HEMELB_LATTICE



class Normal_LBGK_SBB_Nash
{
public:
  typedef lattices:: HEMELB_LATTICE LatticeType;
  typedef typename collisions::Normal<kernels::LBGK<LatticeType>> CollisionType;
  typedef typename streamers::SimpleBounceBackDelegate<CollisionType> WallLinkType;
  typedef typename streamers::NashZerothOrderPressureDelegate<CollisionType> IoletLinkType;

  typedef typename streamers::StreamerTypeFactory<
    CollisionType,
    WallLinkType,
    IoletLinkType
  > Type;
};



__global__
void Normal_LBGK_SBB_Nash_StreamAndCollide(
  site_t firstIndex,
  site_t siteCount,
  distribn_t lbmParams_tau,
  distribn_t lbmParams_omega,
  const iolets::InOutLetCosineGPU* inlets,
  const iolets::InOutLetCosineGPU* outlets,
  const site_t* streamingIndices,
  const geometry::SiteData* siteData,
  const distribn_t* fOld,
  distribn_t* fNew,
  site_t totalSiteCount,
  unsigned long timeStep
)
{
  site_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if ( i >= siteCount )
  {
    return;
  }

  site_t siteIndex = firstIndex + i;
  auto& site = siteData[siteIndex];

  // initialize hydroVars
  distribn_t f_old_j;
  distribn_t f_new_j;
  distribn_t density = 0.0;
  double3 momentum = make_double3(0.0, 0.0, 0.0);

  for ( Direction j = 0; j < DmQn::NUMVECTORS; ++j )
  {
    // copy fOld[i, j] to local memory
    f_old_j = fOld[j * totalSiteCount + siteIndex];

    // Normal::DoCalculatePreCollision()
    // LBGK::DoCalculateDensityMomentumFeq()
    // Lattice::CalculateDensityAndMomentum()
    density += f_old_j;
    momentum.x += DmQn::CXD[j] * f_old_j;
    momentum.y += DmQn::CYD[j] * f_old_j;
    momentum.z += DmQn::CZD[j] * f_old_j;
  }

  // Lattice::CalculateFeq()
  const distribn_t density_1 = 1. / density;
  const distribn_t momentumMagnitudeSquared =
      momentum.x * momentum.x
      + momentum.y * momentum.y
      + momentum.z * momentum.z;

  for ( Direction j = 0; j < DmQn::NUMVECTORS; ++j )
  {
    if ( site.HasIolet(j) )
    {
      // NashZerothOrderPressureDelegate::StreamLink()
      // get iolet
      auto& iolet = (site.GetSiteType() == geometry::INLET_TYPE)
        ? inlets[site.GetIoletId()]
        : outlets[site.GetIoletId()];

      // get density at the iolet
      distribn_t ioletDensity = iolet.GetDensity(timeStep);

      // compute momentum at the iolet
      distribn_t component =
          momentum.x * iolet.normal.x
          + momentum.y * iolet.normal.y
          + momentum.z * iolet.normal.z;

      component *= ioletDensity / density;

      double3 ioletMomentum;
      ioletMomentum.x = iolet.normal.x * component;
      ioletMomentum.y = iolet.normal.y * component;
      ioletMomentum.z = iolet.normal.z * component;

      // compute f_eq at the iolet
      // Lattice::CalculateFeq()
      const distribn_t ioletDensity_1 = 1. / ioletDensity;
      const distribn_t momentumMagnitudeSquared =
          ioletMomentum.x * ioletMomentum.x
          + ioletMomentum.y * ioletMomentum.y
          + ioletMomentum.z * ioletMomentum.z;

      Direction jj = DmQn::INVERSEDIRECTIONS[j];
      const distribn_t mom_dot_ei =
          DmQn::CXD[jj] * ioletMomentum.x
          + DmQn::CYD[jj] * ioletMomentum.y
          + DmQn::CZD[jj] * ioletMomentum.z;

      f_new_j = DmQn::EQMWEIGHTS[jj]
          * (ioletDensity
              - (3. / 2.) * ioletDensity_1 * momentumMagnitudeSquared
              + (9. / 2.) * ioletDensity_1 * mom_dot_ei * mom_dot_ei
              + 3. * mom_dot_ei);
    }
    else
    {
      // copy fOld[i, j] to local memory
      f_old_j = fOld[j * totalSiteCount + siteIndex];

      // Lattice::CalculateFeq()
      const distribn_t mom_dot_ei =
          DmQn::CXD[j] * momentum.x
          + DmQn::CYD[j] * momentum.y
          + DmQn::CZD[j] * momentum.z;

      f_new_j = DmQn::EQMWEIGHTS[j]
          * (density
              - (3. / 2.) * density_1 * momentumMagnitudeSquared
              + (9. / 2.) * density_1 * mom_dot_ei * mom_dot_ei
              + 3. * mom_dot_ei);

      // Normal::DoCollide()
      // LBGK::DoCollide()
      f_new_j = f_old_j + lbmParams_omega * (f_old_j - f_new_j);
    }

    // stream f_new_j to pre-determined output location
    site_t outputIndex = streamingIndices[j * totalSiteCount + siteIndex];

    fNew[outputIndex] = f_new_j;
  }
}



template<>
void Normal_LBGK_SBB_Nash::Type::StreamAndCollideGPU(
  const site_t firstIndex,
  const site_t siteCount,
  const lb::LbmParameters* lbmParams,
  geometry::LatticeData* latDat,
  lb::SimulationState* simState,
  const iolets::InOutLetCosineGPU* inlets,
  const iolets::InOutLetCosineGPU* outlets,
  int blockSize
)
{
  if ( siteCount == 0 )
  {
    return;
  }

  int gridSize = (siteCount + blockSize - 1) / blockSize;

  Normal_LBGK_SBB_Nash_StreamAndCollide<<<gridSize, blockSize>>>(
    firstIndex,
    siteCount,
    lbmParams->GetTau(),
    lbmParams->GetOmega(),
    inlets,
    outlets,
    latDat->GetStreamingIndicesGPU(),
    latDat->GetSiteDataGPU(),
    latDat->GetFOldGPU(0),
    latDat->GetFNewGPU(0),
    latDat->GetLocalFluidSiteCount(),
    simState->Get0IndexedTimeStep()
  );
  CUDA_SAFE_CALL(hipGetLastError());
}

#include "hip/hip_runtime.h"

// This file is part of HemeLB and is Copyright (C)
// the HemeLB team and/or their institutions, as detailed in the
// file AUTHORS. This software is provided under the terms of the
// license in the file LICENSE.

#include "lb/iolets/InOutLetCosine.cuh"
#include "lb/lattices/D3Q15.cuh"



namespace hemelb {
namespace lb {
namespace streamers {



#define D3Q15 lattices::D3Q15



// geometry/SiteType.h
enum site_type_t
{
  SOLID_TYPE = 0U,
  FLUID_TYPE = 1U,
  INLET_TYPE = 2U,
  OUTLET_TYPE = 3U
};



// geometry/SiteDataBare.h
typedef struct
{
  unsigned wallIntersection;
  unsigned ioletIntersection;
  site_type_t type;
  int ioletId;
} site_data_t;



__device__ bool Site_HasIolet(unsigned ioletIntersection, int direction)
{
  unsigned mask = 1U << (direction - 1);
  return ((ioletIntersection & mask) != 0) && (direction > 0);
}



__device__ bool Site_HasWall(unsigned wallIntersection, int direction)
{
  unsigned mask = 1U << (direction - 1);
  return ((wallIntersection & mask) != 0) && (direction > 0);
}



// lb/lattices/Lattice.h
__device__ void Lattice_CalculateFeq(const distribn_t& density, const double3& momentum, distribn_t* f_eq)
{
  const distribn_t density_1 = 1. / density;
  const distribn_t momentumMagnitudeSquared =
      momentum.x * momentum.x
      + momentum.y * momentum.y
      + momentum.z * momentum.z;

  for ( int j = 0; j < D3Q15::NUMVECTORS; ++j )
  {
    const distribn_t mom_dot_ei =
        D3Q15::CXD[j] * momentum.x
        + D3Q15::CYD[j] * momentum.y
        + D3Q15::CZD[j] * momentum.z;

    f_eq[j] = D3Q15::EQMWEIGHTS[j]
        * (density
            - (3. / 2.) * momentumMagnitudeSquared * density_1
            + (9. / 2.) * density_1 * mom_dot_ei * mom_dot_ei
            + 3. * mom_dot_ei);
  }
}



__global__ void DoStreamAndCollideKernel(
  site_t firstIndex,
  site_t siteCount,
  distribn_t lbmParams_tau,
  distribn_t lbmParams_omega,
  const iolets::InOutLetCosineGPU* inlets,
  const iolets::InOutLetCosineGPU* outlets,
  const site_t* neighbourIndices,
  const site_data_t* siteData,
  const distribn_t* fOld,
  distribn_t* fNew,
  unsigned long timeStep
)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if ( i >= siteCount )
  {
    return;
  }

  site_t siteIndex = firstIndex + i;

  // initialize hydroVars
  distribn_t f[D3Q15::NUMVECTORS];
  distribn_t density;
  double3 momentum;
  double3 velocity;
  distribn_t f_eq[D3Q15::NUMVECTORS];
  distribn_t* f_neq = f_eq;
  distribn_t* f_post = f_eq;

  // copy fOld to local memory
  memcpy(&f[0], &fOld[siteIndex * D3Q15::NUMVECTORS], D3Q15::NUMVECTORS * sizeof(distribn_t));

  // collider.CalculatePreCollision() (collider = Normal, kernel = LBGK)

  // Lattice::CalculateDensityMomentumFEq()
  density = 0.0;
  momentum.x = 0.0;
  momentum.y = 0.0;
  momentum.z = 0.0;

  for ( int j = 0; j < D3Q15::NUMVECTORS; ++j )
  {
    density += f[j];
    momentum.x += D3Q15::CXD[j] * f[j];
    momentum.y += D3Q15::CYD[j] * f[j];
    momentum.z += D3Q15::CZD[j] * f[j];
  }

  velocity.x = momentum.x / density;
  velocity.y = momentum.y / density;
  velocity.z = momentum.z / density;

  Lattice_CalculateFeq(density, momentum, f_eq);

  // LBGK::DoCalculateDensityMomentumFeq()
  for ( int j = 0; j < D3Q15::NUMVECTORS; ++j )
  {
    f_neq[j] = f[j] - f_eq[j];
  }

  // collider.Collide()

  // LBGK::DoCollide()
  for ( int j = 0; j < D3Q15::NUMVECTORS; ++j )
  {
    f_post[j] = f[j] + f_neq[j] * lbmParams_omega;
  }

  // perform streaming
  site_data_t site = siteData[siteIndex];

  for ( int j = 0; j < D3Q15::NUMVECTORS; ++j )
  {
    if ( Site_HasIolet(site.ioletIntersection, j) )
    {
      // get iolet
      iolets::InOutLetCosineGPU iolet = (site.type == INLET_TYPE)
        ? inlets[site.ioletId]
        : outlets[site.ioletId];

      // get density at the iolet
      distribn_t ghost_density = iolet.GetDensity(timeStep);

      // compute momentum at the iolet
      distribn_t component =
          velocity.x * iolet.normal.x
          + velocity.y * iolet.normal.y
          + velocity.z * iolet.normal.z;

      double3 ghost_momentum;
      ghost_momentum.x = iolet.normal.x * component * ghost_density;
      ghost_momentum.y = iolet.normal.y * component * ghost_density;
      ghost_momentum.z = iolet.normal.z * component * ghost_density;

      // compute f_eq at the iolet
      distribn_t ghost_f_eq[D3Q15::NUMVECTORS];

      Lattice_CalculateFeq(ghost_density, ghost_momentum, ghost_f_eq);

      int outIndex = siteIndex * D3Q15::NUMVECTORS + D3Q15::INVERSEDIRECTIONS[j];
      fNew[outIndex] = ghost_f_eq[D3Q15::INVERSEDIRECTIONS[j]];
    }
    else if ( Site_HasWall(site.wallIntersection, j) )
    {
      int outIndex = siteIndex * D3Q15::NUMVECTORS + D3Q15::INVERSEDIRECTIONS[j];
      fNew[outIndex] = f_post[j];
    }
    else
    {
      int outIndex = neighbourIndices[siteIndex * D3Q15::NUMVECTORS + j];
      fNew[outIndex] = f_post[j];
    }
  }
}



__host__ void DoStreamAndCollideGPU(
  site_t firstIndex,
  site_t siteCount,
  distribn_t lbmParams_tau,
  distribn_t lbmParams_omega,
  const iolets::InOutLetCosineGPU* inlets,
  const iolets::InOutLetCosineGPU* outlets,
  const site_t* neighbourIndices,
  const void* siteData,
  const distribn_t* fOld,
  distribn_t* fNew,
  unsigned long timeStep
)
{
  const int BLOCK_SIZE = 256;
  const int GRID_SIZE = (siteCount + BLOCK_SIZE - 1) / BLOCK_SIZE;

  DoStreamAndCollideKernel<<<GRID_SIZE, BLOCK_SIZE>>>(
    firstIndex,
    siteCount,
    lbmParams_tau,
    lbmParams_omega,
    inlets,
    outlets,
    neighbourIndices,
    (site_data_t*) siteData,
    fOld,
    fNew,
    timeStep
  );
}



}
}
}

#include "hip/hip_runtime.h"

// This file is part of HemeLB and is Copyright (C)
// the HemeLB team and/or their institutions, as detailed in the
// file AUTHORS. This software is provided under the terms of the
// license in the file LICENSE.

#include "lb/collisions/Collisions.h"
#include "lb/iolets/InOutLetCosine.cuh"
#include "lb/kernels/Kernels.h"
#include "lb/lattices/Lattices.h"
#include "lb/streamers/Streamers.h"

#include "lb/lattices/D3Q15.cuh"
#include "lb/lattices/D3Q19.cuh"
#include "lb/lattices/D3Q27.cuh"



using namespace hemelb;
using namespace hemelb::lb;



#define DmQn lattices::GPU:: HEMELB_LATTICE



// lb/lattices/Lattice.h
__device__ void Lattice_CalculateFeq(
  const distribn_t& density,
  const double3& momentum,
  distribn_t* f_eq)
{
  const distribn_t density_1 = 1. / density;
  const distribn_t momentumMagnitudeSquared =
      momentum.x * momentum.x
      + momentum.y * momentum.y
      + momentum.z * momentum.z;

  for ( Direction j = 0; j < DmQn::NUMVECTORS; ++j )
  {
    const distribn_t mom_dot_ei =
        DmQn::CXD[j] * momentum.x
        + DmQn::CYD[j] * momentum.y
        + DmQn::CZD[j] * momentum.z;

    f_eq[j] = DmQn::EQMWEIGHTS[j]
        * (density
            - (3. / 2.) * density_1 * momentumMagnitudeSquared
            + (9. / 2.) * density_1 * mom_dot_ei * mom_dot_ei
            + 3. * mom_dot_ei);
  }
}



__global__ void Normal_LBGK_SBB_Nash_StreamAndCollide(
  site_t firstIndex,
  site_t siteCount,
  distribn_t lbmParams_tau,
  distribn_t lbmParams_omega,
  const iolets::InOutLetCosineGPU* inlets,
  const iolets::InOutLetCosineGPU* outlets,
  const site_t* neighbourIndices,
  const geometry::SiteData* siteData,
  const distribn_t* fOld,
  distribn_t* fNew,
  unsigned long timeStep
)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if ( i >= siteCount )
  {
    return;
  }

  site_t siteIndex = firstIndex + i;

  // initialize hydroVars
  distribn_t f[DmQn::NUMVECTORS];
  distribn_t density = 0.0;
  double3 momentum = make_double3(0.0, 0.0, 0.0);
  distribn_t f_post[DmQn::NUMVECTORS];

  for ( Direction j = 0; j < DmQn::NUMVECTORS; ++j )
  {
    // copy fOld to local memory
    f[j] = fOld[siteIndex * DmQn::NUMVECTORS + j];

    // Normal::DoCalculatePreCollision()
    // LBGK::DoCalculateDensityMomentumFeq()
    // Lattice::CalculateDensityAndMomentum()
    density += f[j];
    momentum.x += DmQn::CXD[j] * f[j];
    momentum.y += DmQn::CYD[j] * f[j];
    momentum.z += DmQn::CZD[j] * f[j];
  }

  // Lattice::CalculateFeq()
  const distribn_t density_1 = 1. / density;
  const distribn_t momentumMagnitudeSquared =
      momentum.x * momentum.x
      + momentum.y * momentum.y
      + momentum.z * momentum.z;

  for ( Direction j = 0; j < DmQn::NUMVECTORS; ++j )
  {
    const distribn_t mom_dot_ei =
        DmQn::CXD[j] * momentum.x
        + DmQn::CYD[j] * momentum.y
        + DmQn::CZD[j] * momentum.z;

    f_post[j] = DmQn::EQMWEIGHTS[j]
        * (density
            - (3. / 2.) * density_1 * momentumMagnitudeSquared
            + (9. / 2.) * density_1 * mom_dot_ei * mom_dot_ei
            + 3. * mom_dot_ei);

    // Normal::DoCollide()
    // LBGK::DoCollide()
    f_post[j] = f[j] + lbmParams_omega * (f[j] - f_post[j]);
  }

  // perform streaming
  auto& site = siteData[siteIndex];

  for ( Direction j = 0; j < DmQn::NUMVECTORS; ++j )
  {
    if ( site.HasIolet(j) )
    {
      // NashZerothOrderPressureDelegate::StreamLink()
      // get iolet
      auto& iolet = (site.GetSiteType() == geometry::INLET_TYPE)
        ? inlets[site.GetIoletId()]
        : outlets[site.GetIoletId()];

      // get density at the iolet
      distribn_t ioletDensity = iolet.GetDensity(timeStep);

      // compute momentum at the iolet
      distribn_t component =
          (momentum.x / density) * iolet.normal.x
          + (momentum.y / density) * iolet.normal.y
          + (momentum.z / density) * iolet.normal.z;

      double3 ioletMomentum;
      ioletMomentum.x = iolet.normal.x * component * ioletDensity;
      ioletMomentum.y = iolet.normal.y * component * ioletDensity;
      ioletMomentum.z = iolet.normal.z * component * ioletDensity;

      // compute f_eq at the iolet
      distribn_t ioletFeq[DmQn::NUMVECTORS];

      Lattice_CalculateFeq(ioletDensity, ioletMomentum, ioletFeq);

      int outIndex = siteIndex * DmQn::NUMVECTORS + DmQn::INVERSEDIRECTIONS[j];
      fNew[outIndex] = ioletFeq[DmQn::INVERSEDIRECTIONS[j]];
    }
    else if ( site.HasWall(j) )
    {
      // SimpleBounceBackDelegate::StreamLink()
      int outIndex = siteIndex * DmQn::NUMVECTORS + DmQn::INVERSEDIRECTIONS[j];
      fNew[outIndex] = f_post[j];
    }
    else
    {
      // SimpleCollideAndStreamDelegate::StreamLink()
      int outIndex = neighbourIndices[siteIndex * DmQn::NUMVECTORS + j];
      fNew[outIndex] = f_post[j];
    }
  }
}



class Normal_LBGK_SBB_Nash
{
public:
  typedef lattices:: HEMELB_LATTICE LatticeType;
  typedef typename collisions::Normal<kernels::LBGK<LatticeType>> CollisionType;
  typedef typename streamers::SimpleBounceBackDelegate<CollisionType> WallLinkType;
  typedef typename streamers::NashZerothOrderPressureDelegate<CollisionType> IoletLinkType;

  typedef typename streamers::StreamerTypeFactory<
    CollisionType,
    WallLinkType,
    IoletLinkType
  > Type;
};



template<>
void Normal_LBGK_SBB_Nash::Type::StreamAndCollideGPU(
  const site_t firstIndex,
  const site_t siteCount,
  const lb::LbmParameters* lbmParams,
  geometry::LatticeData* latDat,
  lb::SimulationState* simState,
  const iolets::InOutLetCosineGPU* inlets,
  const iolets::InOutLetCosineGPU* outlets
)
{
  if ( siteCount == 0 )
  {
    return;
  }

  const int BLOCK_SIZE = 256;
  const int GRID_SIZE = (siteCount + BLOCK_SIZE - 1) / BLOCK_SIZE;

  Normal_LBGK_SBB_Nash_StreamAndCollide<<<GRID_SIZE, BLOCK_SIZE>>>(
    firstIndex,
    siteCount,
    lbmParams->GetTau(),
    lbmParams->GetOmega(),
    inlets,
    outlets,
    latDat->GetNeighbourIndicesGPU(),
    latDat->GetSiteDataGPU(),
    latDat->GetFOldGPU(0),
    latDat->GetFNewGPU(0),
    simState->Get0IndexedTimeStep()
  );
  CUDA_SAFE_CALL(hipGetLastError());
}

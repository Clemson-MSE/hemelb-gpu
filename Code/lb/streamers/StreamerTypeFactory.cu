
#include <hip/hip_runtime.h>

namespace hemelb {
namespace lb {
namespace streamers {

// type definitions taken from host code
typedef int64_t site_t;
typedef double distribn_t;

// constants (to be implemented as template parameters)
__device__ const int D3Q15_NUMVECTORS = 15;

__device__ const int D3Q15_CX[] = { 0, 1, -1, 0, 0, 0, 0, 1, -1, 1, -1, 1, -1, 1, -1 };
__device__ const int D3Q15_CY[] = { 0, 0, 0, 1, -1, 0, 0, 1, -1, 1, -1, -1, 1, -1, 1 };
__device__ const int D3Q15_CZ[] = { 0, 0, 0, 0, 0, 1, -1, 1, -1, -1, 1, 1, -1, -1, 1 };

__device__ const distribn_t D3Q15_CXD[] = { 0.0, 1.0, -1.0, 0.0,  0.0, 0.0,  0.0, 1.0, -1.0,  1.0, -1.0,  1.0, -1.0,  1.0, -1.0};
__device__ const distribn_t D3Q15_CYD[] = { 0.0, 0.0,  0.0, 1.0, -1.0, 0.0,  0.0, 1.0, -1.0,  1.0, -1.0, -1.0,  1.0, -1.0,  1.0};
__device__ const distribn_t D3Q15_CZD[] = { 0.0, 0.0,  0.0, 0.0,  0.0, 1.0, -1.0, 1.0, -1.0, -1.0,  1.0,  1.0, -1.0, -1.0,  1.0};

__device__ const distribn_t D3Q15_EQMWEIGHTS[] = {
  2.0 / 9.0,
  1.0 / 9.0,
  1.0 / 9.0,
  1.0 / 9.0,
  1.0 / 9.0,
  1.0 / 9.0,
  1.0 / 9.0,
  1.0 / 72.0,
  1.0 / 72.0,
  1.0 / 72.0,
  1.0 / 72.0,
  1.0 / 72.0,
  1.0 / 72.0,
  1.0 / 72.0,
  1.0 / 72.0
};

__device__ const int D3Q15_INVERSEDIRECTIONS[] = { 0, 2, 1, 4, 3, 6, 5, 8, 7, 10, 9, 12, 11, 14, 13 };



__device__ bool Site_HasWall(unsigned wallIntersection, int direction)
{
  unsigned mask = 1U << max(0, direction - 1);
  return (wallIntersection & mask) != 0;
}



__global__ void DoStreamAndCollideKernel(
  site_t firstIndex,
  site_t siteCount,
  distribn_t lbmParams_tau,
  distribn_t lbmParams_omega,
  const site_t* neighbourIndices,
  const unsigned* wallIntersections,
  const distribn_t* fOld,
  distribn_t* fNew
)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if ( i >= siteCount )
  {
    return;
  }

  site_t siteIndex = firstIndex + i;

  // initialize hydroVars
  distribn_t f[D3Q15_NUMVECTORS];
  distribn_t density;
  double3 momentum;
  double3 velocity;
  distribn_t f_eq[D3Q15_NUMVECTORS];
  distribn_t* f_neq = f_eq;
  distribn_t* f_post = f_eq;

  // copy fOld to local memory
  memcpy(&f[0], &fOld[siteIndex * D3Q15_NUMVECTORS], D3Q15_NUMVECTORS * sizeof(distribn_t));

  // collider.CalculatePreCollision() (collider = Normal, kernel = LBGK)

  // Lattice::CalculateDensityMomentumFEq()
  density = 0.0;
  momentum.x = 0.0;
  momentum.y = 0.0;
  momentum.z = 0.0;

  for ( int j = 0; j < D3Q15_NUMVECTORS; ++j )
  {
    density += f[j];
    momentum.x += D3Q15_CXD[j] * f[j];
    momentum.y += D3Q15_CYD[j] * f[j];
    momentum.z += D3Q15_CZD[j] * f[j];
  }

  velocity.x = momentum.x / density;
  velocity.y = momentum.y / density;
  velocity.z = momentum.z / density;

  const distribn_t density_1 = 1. / density;
  const distribn_t momentumMagnitudeSquared =
      momentum.x * momentum.x
      + momentum.y * momentum.y
      + momentum.z * momentum.z;

  for ( int j = 0; j < D3Q15_NUMVECTORS; ++j )
  {
    const distribn_t mom_dot_ei =
        D3Q15_CX[j] * momentum.x
        + D3Q15_CY[j] * momentum.y
        + D3Q15_CZ[j] * momentum.z;

    f_eq[j] = D3Q15_EQMWEIGHTS[j]
        * (density
            - (3. / 2.) * momentumMagnitudeSquared * density_1
            + (9. / 2.) * density_1 * mom_dot_ei * mom_dot_ei
            + 3. * mom_dot_ei);
  }

  // LBGK::DoCalculateDensityMomentumFeq()
  for ( int j = 0; j < D3Q15_NUMVECTORS; ++j )
  {
    f_neq[j] = f[j] - f_eq[j];
  }

  // collider.Collide()

  // LBGK::DoCollide()
  for ( int j = 0; j < D3Q15_NUMVECTORS; ++j )
  {
    f_post[j] = f[j] + f_neq[j] * lbmParams_omega;
  }

  // perform streaming
  for ( int j = 0; j < D3Q15_NUMVECTORS; ++j )
  {
    if ( Site_HasWall(wallIntersections[siteIndex], j) )
    {
      int outIndex = siteIndex * D3Q15_NUMVECTORS + D3Q15_INVERSEDIRECTIONS[j];
      fNew[outIndex] = f_post[j];
    }
    else
    {
      int outIndex = neighbourIndices[siteIndex * D3Q15_NUMVECTORS + j];
      fNew[outIndex] = f_post[j];
    }
  }
}



void DoStreamAndCollideGPU(
  site_t firstIndex,
  site_t siteCount,
  distribn_t lbmParams_tau,
  distribn_t lbmParams_omega,
  const site_t* neighbourIndices,
  const unsigned* wallIntersections,
  const distribn_t* fOld,
  distribn_t* fNew
)
{
  const int BLOCK_SIZE = 256;
  const int GRID_SIZE = (siteCount + BLOCK_SIZE - 1) / BLOCK_SIZE;

  DoStreamAndCollideKernel<<<GRID_SIZE, BLOCK_SIZE>>>(
    firstIndex,
    siteCount,
    lbmParams_tau,
    lbmParams_omega,
    neighbourIndices,
    wallIntersections,
    fOld,
    fNew
  );
}



}
}
}

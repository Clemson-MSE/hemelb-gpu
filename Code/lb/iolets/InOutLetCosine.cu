#include "hip/hip_runtime.h"

// This file is part of HemeLB and is Copyright (C)
// the HemeLB team and/or their institutions, as detailed in the
// file AUTHORS. This software is provided under the terms of the
// license in the file LICENSE.

#include "lb/iolets/InOutLetCosine.cuh"



using namespace hemelb;
using namespace hemelb::lb::iolets;



__device__ distribn_t InOutLetCosineGPU::GetDensity(unsigned long timeStep) const
{
  distribn_t w = 2.0 * M_PI / period;
  distribn_t target = densityMean + densityAmp * cos(w * timeStep + phase);

  if (timeStep >= warmUpLength)
  {
    return target;
  }

  double interpolationFactor = ((double) timeStep) / ((double) warmUpLength);

  return interpolationFactor * target + (1. - interpolationFactor) * minimumSimulationDensity;
}
